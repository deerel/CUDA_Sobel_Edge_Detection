#include "hip/hip_runtime.h"
#include "opencv2\opencv.hpp"
#include "img_cuda.h"
#include "hip/hip_runtime.h"
#include "img_helper.h"
#include ""

#include <stdio.h>

using namespace cv;

__global__ void kernel_grayscale(pixel * img) {


}

__global__ void kernel_gaussian() {

}

__global__ void kernel_sobel() {

}

__global__ void kernel_normalize() {

}

void cuda_edge_detection(Mat * image) {
	pixel * h_image;
	pixel * d_image;
	int elements = (*image).cols * (*image).rows;

	matToArray(image, h_image);

	hipMalloc((void**)&d_image, elements * sizeof(pixel));
	hipMemcpy(d_image, image, elements * sizeof(pixel), hipMemcpyHostToDevice);
	kernel_grayscale << <64, 64 >> >(d_image);

	hipFree(d_image);

}