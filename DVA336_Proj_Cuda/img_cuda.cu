#include "hip/hip_runtime.h"
#include "opencv2\opencv.hpp"
#include "img_cuda.h"
#include "hip/hip_runtime.h"
#include "img_helper.h"
#include ""

#include <stdio.h>

using namespace cv;

#define BLOCKS 512
#define THREADS 256

__global__ void kernel_grayscale(pixel * src, int16_t * dst, const int width, const int height) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	const int n = width * height;
	while (index < n) {
		dst[index] = (int16_t)(src[index].b + src[index].r + src[index].g) / 3;
		index += stride;
	}
}

// Extend this image with a 1 pixel border with value 0;
__global__ void kernel_gaussian(int16_t * src, int16_t * dst, matrix mat, const int width, const int height) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int pixelValue;
	int pixelAcc = 0;
	const int noElements = width * height;

	if (index == 0) {
		printf("MATRIX\n");
		for (int i = 0; i < 3; i++)
		{
			for (int j = 0; j < 3; j++)
			{
				printf("%d,", mat.element[i][j]);
			}
			printf("\n");
		}
		printf("\n");
	}

	if (index == 0) {
		printf("SOURCE SNIPPET\n");
		for (int i = 0; i < 64; i++)
		{
			printf("%d,", src[i]);
		}
		printf("\n");
	}


	//while (index < noElements) {

	//	for (int i = 0; i < 3; i++)
	//	{
	//		for (int j = 0; j < 3; j++)
	//		{
	//			int rowOffset = (i - 1)*width;
	//			int elementOffset = (j - 1);
	//			int pixel_index = index + rowOffset + elementOffset;

	//			pixelAcc += mat.element[i][j] * src[pixel_index];
	//		}
	//	}

	//	dst[index] = pixelAcc / 16;


	//	//dst[index] = src[index];
	//	index += stride;
	//}

	while (index < noElements) {
		if (index > width*2 - 1 && index < width*(height - 2)-1) {
			for (int i = 0; i < 3; i++)
			{
				for (int j = 0; j < 3; j++)
				{
					int rowOffset = (i - 1)*width;
					int elementOffset = (j - 1);
					int pixel_index = index + rowOffset + elementOffset;

					pixelAcc += mat.element[i][j] * src[pixel_index];
					if (index < 3)
						printf("index %d, pixel_index = %d\n", index, index);
				}
			}

			if (index < 16)
				printf("pixelAcc = %d\n", pixelAcc);

		}
		dst[index] = src[index];
		index += stride;
		pixelAcc = 0;
	}

}

__global__ void kernel_sobel() {

}

__global__ void kernel_normalize() {

}

void cuda_edge_detection(int16_t * src, Mat * image) {
	pixel * h_src_image;
	int16_t * h_dst_image;
	matrix matrix;
	pixel * d_src_image;
	int16_t * d_dst_image;
	int16_t * d_result_image;

	const int width = image->cols;
	const int height = image->rows;

	const int elements = width * height;
	const int ext_elements = (width + 1) * (height + 1);

	h_src_image = (pixel *)malloc(elements * sizeof(pixel));
	h_dst_image = (int16_t *)malloc(elements * sizeof(int16_t));

	matToArray(image,h_src_image);

	hipMalloc((void**)&d_src_image, elements * sizeof(pixel));
	hipMalloc((void**)&d_dst_image, ext_elements * sizeof(int16_t));
	hipMalloc((void**)&d_result_image, ext_elements * sizeof(int16_t));

	/* Make grayskale*/
	hipMemcpy(d_src_image, h_src_image, elements * sizeof(pixel), hipMemcpyHostToDevice);
	kernel_grayscale << <BLOCKS, THREADS >> >(d_src_image, d_dst_image, widht, height);
	hipDeviceSynchronize();

	/* Gaussian Blur */
	getGaussianKernel(&matrix);
	kernel_gaussian << <BLOCKS, THREADS >> >(d_dst_image, d_result_image, matrix, width, height);
	hipDeviceSynchronize();

	hipMemcpy(src, d_result_image, elements * sizeof(int16_t), hipMemcpyDeviceToHost);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
	}

	hipFree(d_src_image);
	hipFree(d_dst_image);
	free(h_src_image);
	free(h_dst_image);

}